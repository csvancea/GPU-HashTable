#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	int* myRandomAllocation = nullptr;
	glbGpuAllocator->_cudaMalloc((void **) &myRandomAllocation, 16 * sizeof(int));
	glbGpuAllocator->_cudaFree(myRandomAllocation);
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	return false;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	return NULL;
}
